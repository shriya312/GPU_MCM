#include "hip/hip_runtime.h"
#include <stdio.h>
#include <getopt.h>
#include <string>
#include <cstring>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "MCpi.h"
#define THREADS 128

__device__ inline void rand_g (int &result) {
	int thid = threadIdx.x;
	hiprandState_t state;
	hiprand_init (clock64(),thid,0,&state);
	result = hiprand(&state)%RAND_MAX;
	if (result < 0) result = -result;
	//printf (" %d ", result);	
}

__global__ void monteCarlopi (int *din, int length) {
	
	 __shared__ int sm[2*THREADS];

	int thid = threadIdx.x;
	int t = threadIdx.x + blockIdx.x*blockDim.x;
	int a = 2*thid;
	int b = 2*thid + 1;

	double x,y;
	int x1,y1;
	rand_g(x1);	
	rand_g(y1);	

	if (2*t < length) {
		x = (double)x1/RAND_MAX;
		y = (double) y1/RAND_MAX;
		if ((x*x + y*y) < 1)
			sm[a] = 1; 
		else 
			sm[a] = 0;
	} else {
		sm[a] = 0;
	}
	rand_g(x1);	
	rand_g(y1);
	if ( 2*t + 1 < length) {	
		x = (double) x1/RAND_MAX;
		y = (double) y1/RAND_MAX;
		if ((x*x + y*y) < 1)
			sm[b] = 1;
		else
			sm[b] = 0; 
	} else {
		sm[b] = 0;
	}
	// wait for all threads to finish

	//__syncthreads();
	//if (thid == 0) {
	////	printf (" rand %d %f ",  x1, x);
	//	for (int i =0;i<2*THREADS;i++)
	//		printf("%d,",sm[i]);

	//}
	//__syncthreads();

	// find sum of points inside the rectangle using sum function

	sm[a] += sm[b]; // 128 elements
	__syncthreads();
	
	int idx;

	if (thid < 64) {  // 64 elements
		idx = thid * 4;	
		sm[idx] += sm[idx+2];
	}
 	__syncthreads();	
	
	if (thid < 32) { // 32 elements
		idx = thid*8;
		sm[idx] += sm[idx + 4]; 
		
	}
	__syncthreads();

	if (thid < 16) { // 16 elements
		idx = thid*16;
		sm[idx] += sm[idx+8];
	}  
 	__syncthreads();	

	if (thid < 8) { // 8 elements
		idx = thid*32;
		sm[idx] += sm[idx+16];
	}  
 	__syncthreads();	
	
	if (thid < 4) { // 4 elements
		idx = thid*64;
		sm[idx] += sm[idx+32];
	}  
 	__syncthreads();	
		
	if (thid < 2) { // 2 elements
		idx = thid*128;
		sm[idx] += sm[idx+64];
	}  
 	__syncthreads();	

	if (thid == 0) { //  1 elements 
		sm[idx] += sm[idx+128];
		din[blockIdx.x] = sm[a];
		//printf ("sum : %d \n ", sm[idx]);
//		printf (" sum %d", sm[a]);	
	}

}


__global__ void prefixSum ( int *d_in , int *temp, int length) {


	__shared__ int sm[2*THREADS];
	int thid = threadIdx.x;
	int t = blockIdx.x*blockDim.x + threadIdx.x;
	int a = 2*thid;
	int b = 2*thid + 1;
	int idx;
	// find sum of points inside the rectangle using sum function

	if (a < length) 
		sm[a] = d_in[2*t];	
	else
		sm[a] = 0;
	if (b < length) 
		sm[b] = d_in[2*t+1];
	else
		sm[b] = 0;

	sm[a] += sm[b]; // 128 elements
	__syncthreads();
	

	if (thid < 64) {  // 64 elements
		idx = thid * 4;	
		sm[idx] += sm[idx+2];
	}
 	__syncthreads();	
	
	if (thid < 32) { // 32 elements
		idx = thid*8;
		sm[idx] += sm[idx + 4]; 
		
	}
 	__syncthreads();	

	if (thid < 16) { // 16 elements
		idx = thid*16;
		sm[idx] += sm[idx+8];
	}  
 	__syncthreads();	

	if (thid < 8) { // 8 elements
		idx = thid*32;
		sm[idx] += sm[idx+16];
	}  
 	__syncthreads();	
	
	if (thid < 4) { // 4 elements
		idx = thid*64;
		sm[idx] += sm[idx+32];
	}  
 	__syncthreads();	
		
	if (thid < 2) { // 2 elements
		idx = thid*128;
		sm[idx] += sm[idx+64];
	}  
 	__syncthreads();	

	if (thid == 0) { //  1 elements 
		sm[idx] += sm[idx+128];
		temp[blockIdx.x] = sm[idx];
		//printf ("sum p : %d \n ", sm[idx]);
	}

}


__global__ void copy_kernel (int *out , int *in, int length) {
	int t = threadIdx.x + blockIdx.x * blockDim.x;
	if (2*t < length) 
		out[2*t] = in[2*t];
	if ((2*t + 1) < length)
		out[2*t + 1] = in[2*t + 1];

}

void monteCarlopi(int num_blocks, int length, double & pi_val) {

	// store the count in global array
	int *d_in, *temp;
	int  i = 0;
	hipMalloc((void**)&d_in, num_blocks*sizeof(int));
	hipMalloc((void**)&temp, num_blocks*sizeof(int));
	//printf ("num blocks %d ", num_blocks);
	monteCarlopi<<<num_blocks,THREADS>>>(d_in, length);
	if (num_blocks == 1) {
	}
	else {
		int num_blocks1 = num_blocks/(2*THREADS);
		if (num_blocks1 < 1) num_blocks1 = 1;
		while (num_blocks > 1) {
			if (i%2 == 0)  
				prefixSum<<<num_blocks1,THREADS>>>(d_in, temp, num_blocks);
			else 
				prefixSum<<<num_blocks1,THREADS>>>(temp, d_in, num_blocks);
			i++;
			num_blocks = num_blocks1;
			num_blocks1 = num_blocks/(2*THREADS);
			if (num_blocks1 < 1) num_blocks1 = 1;
			//copy_kernel<<<num_blocks1, THREADS>>> (d_in, temp, num_blocks);
		}
	}
	int total_sum;
	if (i%2 == 1)
		hipMemcpy(&total_sum, &temp[0], sizeof(int), hipMemcpyDeviceToHost );
	else
		hipMemcpy(&total_sum, &d_in[0], sizeof(int), hipMemcpyDeviceToHost );
	//printf ("total sum : %d ", total_sum);	
	pi_val = (double) 4*total_sum/ double (length);	
	hipFree(d_in);
}


double cudaMC_pi(int length)
{
	double pi_val;
	int num_blocks = length/(2*THREADS);

	monteCarlopi(num_blocks,length,pi_val); 

	printf ("Value of pi : %f\n ", pi_val);	
	
	return pi_val; 
}
